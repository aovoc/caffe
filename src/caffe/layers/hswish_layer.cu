#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/hswish_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void HSwishForward(const int n, const Dtype* in, Dtype* out/*, Dtype negative_slope*/) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > Dtype(-3.) ? in[index] : 0;
    out[index] = out[index]  < Dtype(3.) ? out[index] : Dtype(6.);
    out[index] = in[index] * out[index] /  Dtype(6.);
  }
}

template <typename Dtype>
void HSwishLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  //Dtype negative_slope = this->layer_param_.hswish_param().negative_slope();
  // NOLINT_NEXT_LINE(whitespace/operators)
  HSwishForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);//, negative_slope);
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

template <typename Dtype>
__global__ void HSwishBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff/*, Dtype negative_slope*/) {
  CUDA_KERNEL_LOOP(index, n) {

    out_diff[index] = in_diff[index] / Dtype(6.)  * \
    ((((in_data[index]+ Dtype(3.)) * (in_data[index]+ Dtype(3.) > 0))*(in_data[index] < Dtype(3.))) + ( Dtype(6.) * (in_data[index] >= Dtype(3.))));
     //std::min(std::max(in_data[index] + Dtype(3.), Dtype(0)),Dtype(6.))  + (in_diff[index] * in_data[index] / Dtype(6.) * (in_data[index]+ Dtype(3.) > //0))*(in_data[index] < Dtype(3.));

//    out_diff[index] = in_diff[index] * ((in_data[index] > 0)        + (in_data[index] <= 0) * negative_slope) * (in_data[index] < Dtype(6.));

  }
}

template <typename Dtype>
void HSwishLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    //Dtype negative_slope = this->layer_param_.hswish_param().negative_slope();
    // NOLINT_NEXT_LINE(whitespace/operators)
    HSwishBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff);//, negative_slope);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(HSwishLayer);


}  // namespace caffe
